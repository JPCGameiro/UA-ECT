#include "hip/hip_runtime.h"
//
// Tomás Oliveira e Silva, November 2017
//

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "common.h"
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// program configuration
//

#ifndef SECTOR_SIZE
# define SECTOR_SIZE  512
#endif
#ifndef N_SECTORS
# define N_SECTORS    (1 << 21)  // can go as high as (1 << 21)
#endif

static void modify_sector_cpu_kernel (unsigned int *sector_data, unsigned int sector_number, unsigned int n_sectors,
                                      unsigned int sector_size);
__global__ static void modify_sector_cuda_kernel (unsigned int * __restrict__ sector_data, unsigned int * __restrict__ sector_number,
                                                  unsigned int n_sectors, unsigned int sector_size);
static double get_delta_time(void);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Main program
//

int main (int argc, char **argv)
{
  printf("%s Starting...\n", argv[0]);
  if (sizeof (unsigned int) != (size_t) 4)
     return 1; // fail with prejudice if an integer does not have 4 bytes

  // set up device
  int dev = 0;
  int i;

  hipDeviceProp_t deviceProp;
  CHECK (hipGetDeviceProperties (&deviceProp, dev));
  printf("Using Device %d: %s\n", dev, deviceProp.name);
  CHECK (hipSetDevice (dev));

  // create memory areas in host and device memory where the disk sectors data and sector numbers will be stored
  size_t sector_data_size;
  size_t sector_number_size;
  unsigned int *host_sector_data, *host_sector_number;
  unsigned int *device_sector_data, *device_sector_number;

  sector_data_size = (size_t) N_SECTORS * (size_t) SECTOR_SIZE;
  sector_number_size = (size_t) N_SECTORS * sizeof (unsigned int);
  if ((sector_data_size + sector_number_size) > (size_t) 1.3e9)
     { fprintf (stderr,"The GTX 480 cannot handle more than 1.5GiB of memory!\n");
       exit (1);
     }
  printf ("Total sector data size: %lu\n", sector_data_size);
  printf ("Total sector numbers data size: %lu\n", sector_number_size);

  host_sector_data = (unsigned int *) malloc (sector_data_size);
  host_sector_number = (unsigned int *) malloc (sector_number_size);
  CHECK (hipMalloc ((void **) &device_sector_data, sector_data_size));
  CHECK (hipMalloc ((void **) &device_sector_number, sector_number_size));

  // initialize the host data
  (void) get_delta_time ();
  srand(0xCCE2021);
  for (i = 0; i < (int) (sector_data_size / (int) sizeof(unsigned int)); i++)
    host_sector_data[i] = 108584447u * (unsigned int) i; // "pseudo-random" data (faster than using the rand() function)
  for(i = 0; i < (int) (sector_number_size / (int)sizeof(unsigned int)); i++)
    host_sector_number[i] = (rand () & 0xFFFF) | ((rand () & 0xFFFF) << 16);
  printf ("The initialization of host data took %.3e seconds\n",get_delta_time ());

  // copy the host data to the device memory
  (void) get_delta_time ();
  CHECK (hipMemcpy (device_sector_data, host_sector_data, sector_data_size, hipMemcpyHostToDevice));
  CHECK (hipMemcpy (device_sector_number, host_sector_number, sector_number_size, hipMemcpyHostToDevice));
  printf ("The transfer of %ld bytes from the host to the device took %.3e seconds\n",
          (long) sector_data_size + (long) sector_number_size, get_delta_time ());

  // run the computational kernel
  // as an example, N_SECTORS threads are launched where each thread deals with one sector
  unsigned int gridDimX,gridDimY,gridDimZ,blockDimX,blockDimY,blockDimZ;
  int n_sectors, sector_size;

  n_sectors = N_SECTORS;
  sector_size = SECTOR_SIZE;
  blockDimX = 1 << 0;                                             // optimize!
  blockDimY = 1 << 0;                                             // optimize!
  blockDimZ = 1 << 0;                                             // do not change!
  gridDimX = 1 << 21;                                             // optimize!
  gridDimY = 1 << 0;                                              // optimize!
  gridDimZ = 1 << 0;                                              // do not change!

  dim3 grid (gridDimX, gridDimY, gridDimZ);
  dim3 block (blockDimX, blockDimY, blockDimZ);

  if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != n_sectors)
     { printf ("Wrong configuration!\n");
       return 1;
     }
  (void) get_delta_time ();
  modify_sector_cuda_kernel <<<grid, block>>> (device_sector_data, device_sector_number, n_sectors, sector_size);
  CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
  CHECK (hipGetLastError ());                                 // check for kernel errors
  printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
         gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

  // copy kernel result back to host side
  unsigned int *modified_device_sector_data;

  modified_device_sector_data = (unsigned int *) malloc (sector_data_size);
  CHECK (hipMemcpy (modified_device_sector_data, device_sector_data, sector_data_size, hipMemcpyDeviceToHost));
  printf ("The transfer of %ld bytes from the device to the host took %.3e seconds\n",
          (long) sector_data_size, get_delta_time ());

  // compute the modified sector data on the CPU
  (void) get_delta_time ();
  for (i = 0; i < N_SECTORS; i++)
    modify_sector_cpu_kernel (&host_sector_data[i*SECTOR_SIZE/(sizeof (unsigned int))], host_sector_number[i], n_sectors, sector_size);
  printf("The cpu kernel took %.3e seconds to run (single core)\n",get_delta_time ());

  // compare
  for(i = 0; i < (int) sector_data_size / (int) sizeof (unsigned int); i++)
    if (host_sector_data[i] != modified_device_sector_data[i])
       { int sector_words = sector_size / (int) sizeof (unsigned int);

         printf ("Mismatch in sector %d, word %d\n", i / sector_words, i % sector_words);
         exit(1);
       }
  printf ("All is well!\n");

  // free device global memory
  CHECK (hipFree (device_sector_data));
  CHECK (hipFree (device_sector_number));

  // free host memory
  free (host_sector_data);
  free (host_sector_number);
  free (modified_device_sector_data);

  // reset device
  CHECK (hipDeviceReset ());

  return 0;
}

static void modify_sector_cpu_kernel (unsigned int *sector_data, unsigned int sector_number, unsigned int n_sectors,
                                      unsigned int sector_size)
{
  unsigned int x, i, a, c, n_words;

  // convert the sector size into number of 4-byte words (it is assumed that sizeof(unsigned int) = 4)
  n_words = sector_size / 4u;

  // initialize the linear congruencial pseudo-random number generator
  // (section 3.2.1.2 of The Art of Computer Programming presents the theory behind the restrictions on a and c)
  i = sector_number;                          // get the sector number
  a = 0xCCE00001u ^ ((i & 0x0F0F0F0Fu) << 2); // a must be a multiple of 4 plus 1
  c = 0x00CCE001u ^ ((i & 0xF0F0F0F0u) >> 3); // c must be odd
  x = 0xCCE02021u;                            // initial state

 // modify the sector data
  for (i = 0u; i < n_words; i++)
  { x = a * x + c;                            // update the pseudo-random generator state
    sector_data[i] ^= x;                      // modify the sector data
  }
}

__global__ static void modify_sector_cuda_kernel (unsigned int * __restrict__ sector_data, unsigned int * __restrict__ sector_number,
                                           unsigned int n_sectors, unsigned int sector_size)
{
  unsigned int x, y, idx, i, a, c, n_words;

  // compute the thread number
  x = (unsigned int) threadIdx.x + (unsigned int) blockDim.x * (unsigned int) blockIdx.x;
  y = (unsigned int) threadIdx.y + (unsigned int) blockDim.y * (unsigned int) blockIdx.y;
  idx = (unsigned int) blockDim.x * (unsigned int) gridDim.x * y + x;
  if (idx >= n_sectors)
     return;                                             // safety precaution

  // convert the sector size into number of 4-byte words (it is assumed that sizeof(unsigned int) = 4)
  // and define boundaries
  n_words = sector_size / 4u;

  // adjust pointers
  sector_data += n_words * idx;
  sector_number += idx;

  // initialize the linear congruencial pseudo-random number generator
  // (section 3.2.1.2 of The Art of Computer Programming presents the theory behind the restrictions on a and c)
  i = sector_number[0];                                  // get the sector number
  a = 0xCCE00001u ^ ((i & 0x0F0F0F0Fu) << 2);            // a must be a multiple of 4 plus 1
  c = 0x00CCE001u ^ ((i & 0xF0F0F0F0u) >> 3);            // c must be odd
  x = 0xCCE02021u;                                       // initial state

  // modify the sector data

  for (i = 0u; i < n_words; i++)
  { x = a * x + c;                                       // update the pseudo-random generator state
    sector_data[i] ^= x;
  }
}

static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}
