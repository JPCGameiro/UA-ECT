#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "common.h"
#include <hip/hip_runtime.h>

static double RandomReal(double low, double high); //generate double between -0,5 and 0.5
__global__ static void prod_matrices_gpu(double *a, double *b, double  *c, int N);
static void prod_matrices_cpu(double *h_a, double *h_b, double *h_result, int N);
static double get_delta_time(void); //medir tempos
static void generateMatrix(double *m, int size);

int main(int argc, char **argv) {

    // set up device
    int dev = 0;

    hipDeviceProp_t deviceProp;
    CHECK (hipGetDeviceProperties (&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK (hipSetDevice (dev)); // a gpu que vou utilizar


    int n = 1024; // degree of a square matrix

    // host space
    int nElem = n * n; // number of elements of a square matrix
    int nBytes = nElem * sizeof(double); // matrix storage space in bytes
    double *h_a, *h_b, *h_c, *h_cc; // h_c result of multiplication of matrix a to b in CPU
                                    // h_cc result of multiplication of matrix a to b in GPU
    h_a = (double *)malloc(nBytes);
    h_b = (double *)malloc(nBytes);
    h_c = (double *)malloc(nBytes);
    h_cc = (double *)malloc(nBytes);

    //generate matrix for a and b
    (void) get_delta_time ();
    generateMatrix(h_a,nElem);
    generateMatrix(h_b,nElem);
    printf("Matrix a and b generated on %.3e seconds\n", get_delta_time());

    //reserve memory for gpu
    double *d_a, *d_b, *d_c;
    CHECK(hipMalloc((void **)&d_a, nBytes));
    CHECK(hipMalloc((void **)&d_b, nBytes));
    CHECK(hipMalloc((void **)&d_c, nBytes));

    (void) get_delta_time ();
    //copy matrix to gpu
    CHECK(hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, nBytes, hipMemcpyHostToDevice));
    printf ("The transfer of %d bytes from the host to the device took %.3e seconds\n",
            2  * nBytes, get_delta_time ());

    // run the computational kernel
    // as an example, nElem threads are launched where each thread deals with one multiplication point
    unsigned int gridDimX,gridDimY,gridDimZ,blockDimX,blockDimY,blockDimZ;

    blockDimX = 1;                                             // optimize! // 1 thread
    blockDimY = 1;                                             // optimize!
    blockDimZ = 1;                                             // do not change! // sempre 1
    gridDimX = nElem;                                          // optimize!
    gridDimY = 1;                                              // optimize!
    gridDimZ = 1;                                              // do not change! // sempre 1

    dim3 grid (gridDimX, gridDimY, gridDimZ);
    dim3 block (blockDimX, blockDimY, blockDimZ);

    if ((gridDimX * gridDimY * gridDimZ * blockDimX * blockDimY * blockDimZ) != nElem)
    { printf ("Wrong configuration!\n");
        return 1;
    }

    (void) get_delta_time ();
    prod_matrices_gpu <<<grid, block>>> (d_a, d_b, d_c, n);
    CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish - aguarda que o gpu acabe de executar
    CHECK (hipGetLastError ());                                 // check for kernel errors // por sempre
    printf("The CUDA kernel <<<(%d,%d,%d), (%d,%d,%d)>>> took %.3e seconds to run\n",
           gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, get_delta_time ());

    // copy kernel result back to host side
    CHECK (hipMemcpy (h_cc, d_c, nBytes, hipMemcpyDeviceToHost));
    printf ("The transfer of %d bytes from the device to the host took %.3e seconds\n",
            nBytes, get_delta_time ());

    // free device global memory
    CHECK (hipFree (d_a)); //gpu
    CHECK (hipFree (d_b));
    CHECK (hipFree (d_c)); //gpu

    // reset device
    CHECK (hipDeviceReset ());

    (void) get_delta_time ();
    prod_matrices_cpu(h_a, h_b, h_c, n);
    printf("The cpu kernel took %.3e seconds to run (single core)\n",get_delta_time ());

    // compare
    for(int i = 0; i < nElem; i++)
      if (((fabs (h_c[i]) <= 1.0e-6) && fabs (h_c[i] - h_cc[i]) > 1.0e-6) || (((fabs (h_c[i]) > 1.0e-6) && fabs ((h_c[i] - h_cc[i]) / h_c[i]) > 1.0e-6)))
         { printf ("Mismatch in element (%d, %d): %.6e(GPU) - %.6e(CPU)\n", i / n, i % n, h_cc[i], h_c[i]);
           exit(1);
         }
    printf ("All is well!\n");

    // free host memory
    free (h_a); //cpu
    free (h_b);
    free (h_c);
    free (h_cc);

    return 0;

}

static double RandomReal(double low, double high){
    double d;

    d = (double) rand() / ((double) RAND_MAX + 1);
    return (low + d * (high - low));
}

static void generateMatrix(double *m, int size){
    for (int j = 0; j < size; j++) {
        m[j] = (double) RandomReal(-0.5, 0.5);
    }
}


__global__ static void prod_matrices_gpu(double *a, double *b, double  *c, int N) {

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int idx = gridDim.x * blockDim.x * y + x;
    int row = idx / N;
    int col = idx % N;
    double sum = 0.0;
    if ((row < N) && (col < N)) {
        for (int i = 0; i < N; i++) {
            sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}


static void prod_matrices_cpu(double *h_a, double *h_b, double *h_result, int N) {
    for (int i = 0; i < N ;++i){
        for (int j = 0; j <N; ++j){
            double sum = 0.0;
            for (int h = 0; h < N; ++h){
                sum += h_a[i * N + h] * h_b[h * N + j];
            }
            h_result[i * N + j] = sum;
        }
    }
}


static double get_delta_time(void){
    static struct timespec t0,t1;

    t0 = t1;
    if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
    {
        perror("clock_gettime");
        exit(1);
    }
    return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}
