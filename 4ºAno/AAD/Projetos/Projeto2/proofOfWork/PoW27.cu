#include "hip/hip_runtime.h"
//
// Tomás Oliveira e Silva
//

#ifndef _use_cuda_
# define _use_cuda_  1
#endif

#include <time.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#if _use_cuda_ != 0
#include <hip/hip_runtime.h>
#endif

#include "cMD5.h"

__global__ static void cMD5_cuda(unsigned int * __restrict__ tokens);

//
// measure the amount of time used so far
//

static double real_time(void)
{
  struct timespec t;

  if(clock_gettime(CLOCK_REALTIME,&t) != 0) // to measure CPU time only, use CLOCK_PROCESS_CPUTIME_ID instead of CLOCK_REALTIME
    return -1.0; // clock_gettime() failed so we return an illegal value
  return (double)t.tv_sec + 1.0e-9 * (double)t.tv_nsec;
}

//
// predefined token data (there are exactly 32 tokens)
//

static struct
{
  unsigned int token[14];      // a "random" token
  unsigned int first_solution; // the smallest good value of n_0
}
token_data[32] =
{
  { // token  0 ( 6 rounds)
    {
      0x51D30262u,0x719173EDu,0xEF43D22Du,0x8CC5710Fu,0x527CDDE0u,0xA83834DCu,0x6BA6EC2Cu,
      0xCF11E082u,0x305F69C0u,0xB9AB6DD3u,0x9D378435u,0xD114633Au,0xB5C4FCD9u,0x330476C4u
    },
    0x0029840Bu
  },
  { // token  1 (18 rounds)
    {
      0x75B416CEu,0xBCDACD17u,0x0A859CB7u,0x103AB7EEu,0xAD20DD57u,0x2975EB52u,0x1DE343C2u,
      0x0255E117u,0x61E31F91u,0xDB8D8CEBu,0x67389489u,0x230D1CC4u,0x871E296Fu,0x5C5DEF9Bu
    },
    0x0225A1C1u
  },
  { // token  2 ( 9 rounds)
    {
      0x431442A5u,0xB16AFB4Bu,0x91C925D3u,0x0A36A2C4u,0xB781E9FAu,0x0FEB4DF7u,0x26DBE996u,
      0xB5E8B359u,0x7C61C01Eu,0x282E47B4u,0x5ECB595Fu,0xD6C19F7Cu,0x9D1D9218u,0xD6DD369Eu
    },
    0x0050B4EDu
  },
  { // token  3 ( 7 rounds)
    {
      0x9D100173u,0x8459D49Au,0xE1A2CE0Au,0xF2975D7Cu,0x3EB006CDu,0xA87E40DCu,0x4E5072FEu,
      0xBAB92A63u,0x4AA2BE6Eu,0xF01A4C59u,0xCD31B86Cu,0x4D0524E5u,0xBC5C99F3u,0x799FDBB4u
    },
    0x006BC0B5u
  },
  { // token  4 (12 rounds)
    {
      0xF2AEEC38u,0x0B867BBFu,0x61DD9C8Du,0xAECA6AB4u,0x1F482A3Du,0x712554D1u,0x6C1400FDu,
      0x0394F952u,0x5FD8D7AFu,0x2ABE16D4u,0x642434A0u,0x2BBEF62Cu,0x477D3D29u,0xA0DB9963u
    },
    0x01A18987u
  },
  { // token  5 (14 rounds)
    {
      0x02F1267Au,0xE016A53Du,0x15273346u,0x86AF0971u,0x29247109u,0xC6E671ACu,0x0EF7597Fu,
      0xC5C3A269u,0x8ECF9F9Bu,0x2BBCA6D5u,0xEA441965u,0x6445BECBu,0x9A295769u,0xC51619AFu
    },
    0x008A8E42u
  },
  { // token  6 ( 4 rounds)
    {
      0x30005430u,0xF9D9CE78u,0x25B559C3u,0xECFD12ECu,0x442D7FA8u,0x7477E24Du,0xFB372D72u,
      0x6F0ECF64u,0x16923498u,0x46F69361u,0xAE3D4A43u,0x5467F10Au,0x7443B816u,0xE3C1A9BEu
    },
    0x00775541u
  },
  { // token  7 (14 rounds)
    {
      0x59161F4Au,0xC1977008u,0x105A72D0u,0x8604ADD4u,0x820128EFu,0xF0DDC9D8u,0x1979CBF7u,
      0x8749F2D6u,0x76F43CE4u,0xB82B3D20u,0x39742379u,0x4193DB00u,0x7FF50286u,0x436F3526u
    },
    0x00CDAF11u
  },
  { // token  8 ( 9 rounds)
    {
      0x52004935u,0x5E410AD7u,0xC8249A9Au,0x7FF392B8u,0x932FA520u,0x1DB97617u,0x4237085Au,
      0xD5E681F8u,0xF6691AC3u,0x76630E45u,0xDE46329Du,0x1CCEBF38u,0xEAB24EF4u,0xF162D49Au
    },
    0x00CEB5F0u
  },
  { // token  9 (13 rounds)
    {
      0x22895989u,0x29103D9Fu,0x21A9705Eu,0x665C6211u,0xCE0EA9A0u,0x9E0E32F0u,0x03793061u,
      0x7A73D769u,0x8B2810D0u,0x4622C474u,0xA0354880u,0x6515B7DDu,0xF94FBB8Fu,0x039ECD96u
    },
    0x01BDAD8Bu
  },
  { // token 10 (16 rounds)
    {
      0x7938E1ECu,0x72551990u,0x79600970u,0x98803492u,0x3EEDC68Eu,0x60A4E994u,0x2E0B6563u,
      0xE2C9C4E4u,0x48BA5BE9u,0xCF8AE52Eu,0x0EF05BFCu,0x7AC8902Au,0x4455B2E1u,0xB760358Au
    },
    0x00178BBEu
  },
  { // token 11 (18 rounds)
    {
      0x3187021Eu,0x0F7BAF84u,0x8E8AF950u,0x4F94D05Cu,0xA533A4B3u,0x7DEE17C7u,0x66996110u,
      0xFAFCD6E8u,0x25EC12F3u,0xAC9F39FFu,0x6F394440u,0x3509BF57u,0x6C8BDF39u,0xD4504ADBu
    },
    0x02B17931u
  },
  { // token 12 ( 5 rounds)
    {
      0x10E7F141u,0xAAF4667Du,0x0067C520u,0xF076712Au,0xFFAB5679u,0x0D867EF2u,0x972E3F22u,
      0x7048B2EFu,0xAC3F7196u,0x4A9D5709u,0xB3D02358u,0xD383945Eu,0x273FCC2Bu,0x4DDF51B9u
    },
    0x0021316Au
  },
  { // token 13 (10 rounds)
    {
      0xA71F3356u,0x87433652u,0x439C65B3u,0x89189369u,0xCC068549u,0x9122810Fu,0x0F56EC90u,
      0x0E432AB6u,0xCE04750Cu,0x059A3A44u,0x44208C8Bu,0xE2A92AD7u,0x1C432F82u,0x6E64C438u
    },
    0x022FE046u
  },
  { // token 14 ( 7 rounds)
    {
      0x59C43543u,0xB61F79DDu,0x702F9114u,0xD8D08BF5u,0x51F8A45Fu,0x169AE6EEu,0x6A21AB1Fu,
      0x10F6DCB2u,0x14954A4Au,0x3DD7C5DBu,0x730DACE2u,0x270F00CDu,0xC2D37A7Du,0xBC9AE087u
    },
    0x0027A3ADu
  },
  { // token 15 (10 rounds)
    {
      0xBB35F546u,0x13C3732Eu,0x3C9F3549u,0x155FF657u,0x6B0FC581u,0x888A59AFu,0x0D1A48F8u,
      0x0384615Bu,0xD92EA445u,0x03983264u,0x7AE8B60Bu,0xFC72CD05u,0x7610FD36u,0x39CB181Au
    },
    0x01CF56A9u
  },
  { // token 16 (15 rounds)
    {
      0x3D6ACA4Bu,0x04767B17u,0x81A2234Cu,0x967F7875u,0xD71E46E2u,0x103C16A0u,0x9C5EB117u,
      0x8D5EA1A0u,0x9D393CA1u,0x63FCA6A0u,0xC0B4D6BDu,0x3667788Au,0x2FA35E89u,0x81C58688u
    },
    0x021A63E6u
  },
  { // token 17 (13 rounds)
    {
      0x1EA04D99u,0x5A261FC8u,0xB9B71390u,0xD4EB98A5u,0x73D3D2EFu,0x66B92CC0u,0x48058553u,
      0xA124CC10u,0x83313A68u,0x0DAEEFD5u,0xFF1B2EA6u,0xE90DCE30u,0x1F690EB0u,0x4E016955u
    },
    0x005CCA9Du
  },
  { // token 18 (11 rounds)
    {
      0x37006027u,0x80844129u,0x137985F7u,0x391916F5u,0x41D8839Cu,0x2611A2C4u,0x327EE70Au,
      0x55F7CD39u,0xDB5EF7F8u,0xABCC426Du,0x8B2F0B4Bu,0xFBC1AA3Cu,0xE2D6FB33u,0xAF2AEFE3u
    },
    0x01E303F6u
  },
  { // token 19 (19 rounds)
    {
      0x7809D33Fu,0x17271419u,0xB6DCDE51u,0x1CC909BCu,0x9296673Fu,0xC845EB6Bu,0xA48A80CFu,
      0x54EB77D7u,0x0A001F98u,0xE441F75Eu,0xB891FDE2u,0x5282C91Bu,0x39565641u,0x7C2DA684u
    },
    0x033DFBD7u
  },
  { // token 20 (17 rounds)
    {
      0xD4844EFDu,0x0DC48CE8u,0xD0C3A383u,0x31183CE3u,0x52E9AB95u,0x9BD4F062u,0x9443932Au,
      0x5411A3D9u,0x350FACADu,0xF66F0202u,0x13D63C8Au,0xD8058C40u,0xE6738249u,0x7B3B6DD5u
    },
    0x0263F363u
  },
  { // token 21 ( 6 rounds)
    {
      0xD930A392u,0x704C01BFu,0x521177D8u,0x0C0BFF39u,0x195F5D05u,0xA309FE46u,0xAA2B244Au,
      0xC1A1F646u,0x53871D74u,0xACB93F77u,0x61F02DF5u,0xAB622F1Au,0xF275DB98u,0x66DA2EE1u
    },
    0x0012E2CFu
  },
  { // token 22 ( 8 rounds)
    {
      0xA1FAAB44u,0xE6DAF81Eu,0x87E375C5u,0x5549A6C3u,0xC1F68953u,0x076AFB3Bu,0x27FDCF74u,
      0x90A962B0u,0xF23CD457u,0x7410B702u,0x489979B8u,0x54AA2D20u,0x1902E890u,0x0D4339A9u
    },
    0x00EA11B7u
  },
  { // token 23 ( 4 rounds)
    {
      0x5FE92C40u,0xDBFA42D1u,0xCD63F596u,0xF13D9E41u,0xE813C4BAu,0x6804F010u,0x77874E11u,
      0x8D5D4B14u,0x1901166Eu,0xF0B2A187u,0x95B4B5F2u,0x518949F5u,0x064697E2u,0x7FDAEEE8u
    },
    0x00FF775Fu
  },
  { // token 24 (11 rounds)
    {
      0x54AB9E37u,0xB6AD18DAu,0xB7618A2Du,0x22FEC66Bu,0x00F27F4Cu,0x9D799CBBu,0x857B4006u,
      0x5B249D09u,0xDB4F5392u,0xCF4EA65Fu,0xEA0F93B1u,0xAB387CD2u,0xA879BAB6u,0x4E23BB44u
    },
    0x009AC1F0u
  },
  { // token 25 (17 rounds)
    {
      0xBB70850Du,0xB79A4564u,0x8AD244D5u,0x0F6E8A3Cu,0xB3F36795u,0xFCDDDDB7u,0x6D5C57F3u,
      0x94145839u,0x43C172DDu,0x2B36AD5Fu,0xD7F38692u,0x16C21D94u,0x06966302u,0x6432195Cu
    },
    0x001BD28Du
  },
  { // token 26 (12 rounds)
    {
      0xC2C187C8u,0xD91C8AE7u,0x32A68471u,0xD4361EB8u,0x21DA2DBFu,0x8998C3BDu,0xDBB579EFu,
      0xA44EEE43u,0x785A48C5u,0x0CB4C581u,0x6D0D5ACEu,0xEF78C737u,0xA906D6EBu,0xBBCA5627u
    },
    0x02A29680u
  },
  { // token 27 (15 rounds)
    {
      0xCA13EE1Bu,0xA9D1FB01u,0x2D167038u,0xF9706098u,0x10C80054u,0xC2490677u,0x0A8DD11Bu,
      0xC1F59CB7u,0xB61B4B75u,0x13E42D7Cu,0x3DB18691u,0xE58FDF5Du,0x29A6008Du,0xB559C943u
    },
    0x00AE462Au
  },
  { // token 28 (16 rounds)
    {
      0x9CCB519Cu,0xCABC4875u,0xA73B87DCu,0xB883829Eu,0xFEC68714u,0xB0CA89EBu,0xA8DF8F78u,
      0x136EC645u,0x24688467u,0x35CE2531u,0x8A455729u,0x8AB50655u,0x55A62435u,0x257D5498u
    },
    0x00D02030u
  },
  { // token 29 ( 8 rounds)
    {
      0xAEC0F6B4u,0x03983F15u,0x74C9810Au,0x87EC7EDDu,0x9BA90EACu,0x386F1D3Bu,0x4A544285u,
      0xFFF27A4Cu,0x995AD833u,0x31AE0E73u,0x1E884983u,0x34B0974Eu,0xDAB19664u,0x27DEE285u
    },
    0x00E8492Bu
  },
  { // token 30 ( 5 rounds)
    {
      0xEE532451u,0xCCF0CA5Cu,0x3283EB8Du,0x5FD06EBAu,0x16C0B2DBu,0x77FE5D39u,0x768BC72Au,
      0x8047569Au,0x42D0E875u,0x840D86ADu,0x391A65FCu,0x7AF820F7u,0x3B3235E4u,0x699DEE35u
    },
    0x01255328u
  },
  { // token 31 (19 rounds)
    {
      0x40807F8Fu,0x05216775u,0xF7125AE7u,0x566A3499u,0x8C2AEC9Cu,0x9A5AD1B0u,0x8535B001u,
      0xF75514CFu,0xF81C41D5u,0x0D6A059Eu,0xE698EBC7u,0xF9FF71C7u,0x0A058C12u,0xF4DBFF1Cu
    },
    0x01F7931Cu
  }
};

//
// CPU-only functions
//
// the custom MD5 function (returning only d[0] mod 2^24) and the PoW function (with n_1 always set to 0)
//

static unsigned int cMD5_cpu (unsigned int t[14], unsigned int n[2])
{
  unsigned int m[16],d[4];

  m[ 0] = t[ 0];
  m[ 1] = t[ 1];
  m[ 2] = t[ 2];
  m[ 3] = t[ 3];
  m[ 4] = t[ 4];
  m[ 5] = t[ 5];
  m[ 6] = t[ 6];
  m[ 7] = t[ 7];
  m[ 8] = t[ 8];
  m[ 9] = t[ 9];
  m[10] = t[10];
  m[11] = t[11];
  m[12] = t[12];
  m[13] = t[13];
  m[14] = n[ 0];
  m[15] = n[ 1];
  cMD5 (m, d);
  return d[0] & 0x00FFFFFFu;
}

static unsigned int PoW_cpu (unsigned int t[14])
{
  unsigned int n[2];

  n[1] = 0u; // always 0
  for (n[0] = 0u; n[0] < (1u << 27); n[0]++)
    if (cMD5_cpu (t, n) == 0u)
       return n[0];
  fprintf(stderr," no n_0 found up to 2^27\n");
  exit(1);
}
#include <sys/time.h>

//
// (macro) call a CUDA driver API function and terminate the program if it reports an error
// it can, and should, be used to test the return value of calls
//

#if _use_cuda_ != 0

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

#endif

//
// main program
//

int main (int argc,char **argv)
{
  //
  // token data check (CPU only)
  //

  if ((argc == 2) && (strcmp (argv[1],"-cpu") == 0))
  {
    unsigned int i,n0;
    double t0,t1,t2;

    t0 = t1 = real_time ();
    for (i = 0u; i < 32u; i++)
    {
      fprintf (stderr, "token %2u [%2u rounds]: ", i, 4u + (token_data[i].token[0] & 0x0F));
      n0 = PoW_cpu (&token_data[i].token[0]);
      if(n0 != token_data[i].first_solution)
      {
        fprintf(stderr, " %u is different from %u\n", n0, token_data[i].first_solution);
        exit(1);
      }
      t2 = real_time ();
      fprintf (stderr, "%8u%s,", n0, (n0 >= (1u << 24)) ? " (>= 2^24)" : "          ");
      fprintf (stderr, " done in %6.3fs", t2 - t1);
      fprintf (stderr, " [%6.3fns per round]\n", 1.0e9 * (t2 - t1) / (double) (1u + n0) / (double)(4u + (token_data[i].token[0] & 0x0F)));
      t1 = t2;
    }
    printf ("All is well (all work done in %.3fs)\n", t1 - t0);
    exit (0);
  }

#if _use_cuda_ != 0
  //
  // proof-of-work computation (CPU and CUDA device)
  //

  if((argc == 2) && (strcmp(argv[1],"-cuda") == 0))
  {
    // set up device

    int dev = 0;

    hipDeviceProp_t deviceProp;
    CHECK (hipGetDeviceProperties (&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK (hipSetDevice (dev));

    //
    // create memory areas in host and device memory where the token data and proof-or-work data will be placed
    //

    size_t data_size;
    unsigned int *host_tokens;
    unsigned int *device_tokens;

    data_size = (size_t)32u * (size_t)16 * sizeof(unsigned int); // 32 tokens, each with 16 32-bit words
    host_tokens = (unsigned int *) malloc (data_size);
    CHECK (hipMalloc ((void **) &device_tokens, data_size));

    //
    // thread identification
    //
    // thread x coordinate ............ threadIdx.x + blockDim.x * blockIdx.x
    // thread y coordinate ............ threadIdx.y + blockDim.y * blockIdx.y
    // thread idx (work identifier) ... y + blockDim.y * gridDim.y * x
    // thread token number ............ idx mod 32
    // thread n_0 ..................... floor (idx / 32u)
    //

    unsigned int n_threads_log2 = 5u + 24u; // 32 tokens, 2^24 values of n_0 per token
    static struct
    {
      unsigned int grid_dim_x_log2;  // valid values: 0 to 29
      unsigned int grid_dim_y_log2;  // valid values: 0 to 15
      unsigned int block_dim_x_log2; // valid values: 0 to 10 \ the sum of the two cannot
      unsigned int block_dim_y_log2; // valid values: 0 to 10 /    be larger than 10
    }
    grids[] =
    {
      { 16u, 8u, 8u, 0u }
    };

    //
    // for each launch grid ...
    //

    printf("            GRID      BLOCK\n");
    printf("----------------  ---------\n");
    printf("         X     Y     X    Y      Time\n");
    printf("---------- -----  ---- ----  --------\n");
    fflush(stdout);

    int g;

    for (g = 0; g < (int)(sizeof(grids) / sizeof(grids[0])); g++)
    {
      //
      // get the launch grid dimensions
      //

      unsigned int grid_dim_x,grid_dim_y,block_dim_x,block_dim_y;

      if ((grids[g].grid_dim_x_log2 + grids[g].grid_dim_y_log2 + grids[g].block_dim_x_log2 + grids[g].block_dim_y_log2) != n_threads_log2)
      {
        fprintf(stderr,"grid #%u is bad\n",g);
        exit(1);
      }
      grid_dim_x = 1u << grids[g].grid_dim_x_log2;
      grid_dim_y = 1u << grids[g].grid_dim_y_log2;
      block_dim_x = 1u << grids[g].block_dim_x_log2;
      block_dim_y = 1u << grids[g].block_dim_y_log2;

      //
      // initialize the host data
      //
      int i, j;

      for (i = 0; i < 32; i++)
      {
        for (j = 0; j < 14; j++)
          host_tokens[16 * i + j] = token_data[i].token[j];
        host_tokens[16 * i + 14] = 0xFFFFFFFFu; // not found mark
        host_tokens[16 * i + 15] = 0xFFFFFFFFu; // not found mark
      }

      //
      // copy the host data to device memory
      //

      CHECK (hipMemcpy (device_tokens, host_tokens, data_size, hipMemcpyHostToDevice));

      //
      // run the kernel (set its arguments first)
      //

      dim3 grid (grid_dim_x, grid_dim_y, 1);
      dim3 block (block_dim_x, block_dim_y, 1);

      double t0 = real_time();
      cMD5_cuda <<<grid, block>>> (device_tokens);
      CHECK (hipDeviceSynchronize ());                            // wait for kernel to finish
      CHECK (hipGetLastError ());                                 // check for kernel errors
      double t1 = real_time();
      printf ("%10u %5u  %4u %4u  %8.5f\n", grid_dim_x, grid_dim_y, block_dim_x, block_dim_y, t1 - t0);
      fflush(stdout);

      //
      // copy the buffer form device memory to CPU memory
      //

      CHECK (hipMemcpy (host_tokens, device_tokens, data_size, hipMemcpyDeviceToHost));

      //
      // check data
      //

      for (i = 0; i < 32; i++)
      {
        if ((host_tokens[16 * i + 15] == 0xFFFFFFFFu) && (token_data[i].first_solution < (1u << 24)))
        {
          fprintf (stderr, "PoW(%u token) not finished! (unexpected)\n", i);
          exit (1);
        }
        if (host_tokens[16 * i + 15] != 0xFFFFFFFFu)
           if ((host_tokens[16 * i + 15] != 0u) || (cMD5_cpu (&host_tokens[16 * i], &host_tokens[16 * i + 14]) != 0u))
           {
             fprintf (stderr, "bad PoW(%u token)\n", i);
             exit (1);
           }
      }
    }
    printf ("---------- -----  ---- ----  --------\n");
    fflush(stdout);

    // free device global memory

    CHECK (hipFree (device_tokens));

    // free host memory

    free (host_tokens);

    // reset device

    CHECK (hipDeviceReset ());

    return 0;
  }
#endif

  return 0;
}

//
// cMD5_cuda kernel (each thread computes one custom MD5 function for one specific value of n for a specific token)
//

__global__ static void cMD5_cuda(unsigned int * __restrict__ tokens)
{
  unsigned int idx,m[16],d[4];

  //
  // get a unique work identifier (idx = token_number + 32 * n_0)
  //
  {
    unsigned int x = (unsigned int)threadIdx.x + (unsigned int)blockDim.x * (unsigned int)blockIdx.x;
    unsigned int y = (unsigned int)threadIdx.y + (unsigned int)blockDim.y * (unsigned int)blockIdx.y;
    idx = y + (unsigned int)blockDim.y * (unsigned int)gridDim.y * x;
  }
  //
  // adjust the token_data pointer
  //
  tokens += 16u * (idx % 32u); // each token uses 16 32-bit words (14 for the token, 2 for an eventual return value)
  
  if(tokens[15] == 0u)
    return;
  
  idx /= 32u;                  // now idx is just n_0

  //
  // read the token data
  //
  m[ 0] = tokens[ 0];
  m[ 1] = tokens[ 1];
  m[ 2] = tokens[ 2];
  m[ 3] = tokens[ 3];
  m[ 4] = tokens[ 4];
  m[ 5] = tokens[ 5];
  m[ 6] = tokens[ 6];
  m[ 7] = tokens[ 7];
  m[ 8] = tokens[ 8];
  m[ 9] = tokens[ 9];
  m[10] = tokens[10];
  m[11] = tokens[11];
  m[12] = tokens[12];
  m[13] = tokens[13];
  //
  // put n_1 and n_0 in the m[] array
  //
  m[14] = idx; // n_0
  m[15] = 0u;  // in this implementation n_1 is always 0
  //
  // compute the custom MD5 function
  //
  cMD5(m,d);
  //
  // check result and exit
  //
  if((d[0] & 0x00FFFFFFu) == 0u)
  {                   // got it! since we are interested in ANY solution, if two or more threads find
    tokens[14] = idx; //         good values for n_0, it does not mater which one writes its solution
    tokens[15] = 0u;  //         first, so here we record a solution in the simplest possible way; as
  }                   //         m[15] is always 0, race conditions are irrelevant here
}
